#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256
#define WARP_SIZE (32)

__global__ void reduce0(float *d_input, float *d_output)
{
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = blockIdx.x * blockDim.x + tid;

    for (unsigned int i = 1; i < blockDim.x; i *= 2)
    {
        if (tid % (i * 2) == 0)
        {
            d_input[global_tid] += d_input[global_tid + i];
            __syncthreads();
        }
    }

    if (tid == 0)
    {
        d_output[blockIdx.x] = d_input[global_tid];
    }
}

__global__ void reduce1(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid];
    __syncthreads();

    for (unsigned int i = 1; i < blockDim.x; i *= 2)
    {
        if (tid % (i * 2) == 0)
        {
            shared_mem[tid] += shared_mem[tid + i];
            __syncthreads();
        }
    }

    if (tid == 0)
    {
        d_output[blockIdx.x] = shared_mem[tid];
    }
}

__global__ void reduce2(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid];
    __syncthreads();

    for (unsigned int i = 1; i < blockDim.x; i *= 2)
    {
        if (tid < blockDim.x / (i * 2))
        {
            shared_mem[2 * i * tid] += shared_mem[2 * i * tid + i];
            __syncthreads();
        }
    }

    if (tid == 0)
    {
        d_output[blockIdx.x] = shared_mem[tid];
    }
}

__global__ void reduce3(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid];
    __syncthreads();

    for (unsigned int i = 1; i < blockDim.x; i *= 2)
    {
        if (tid < blockDim.x / (i * 2))
        {
            shared_mem[tid] += shared_mem[tid + blockDim.x / (i * 2)];
            __syncthreads();
        }
    }

    if (tid == 0)
    {
        d_output[blockIdx.x] = shared_mem[tid];
    }
}

__device__ void warpReduce(volatile float *cache, unsigned int tid)
{
    cache[tid] += cache[tid + 32];
    cache[tid] += cache[tid + 16];
    cache[tid] += cache[tid + 8];
    cache[tid] += cache[tid + 4];
    cache[tid] += cache[tid + 2];
    cache[tid] += cache[tid + 1];
}

__global__ void reduce5(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = 2 * blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid] + d_input[global_tid + blockDim.x];
    __syncthreads();

    // Division is not effective, using bit-ops. And we unfold the last loop to reduce the sync time.
    for (unsigned int i = blockDim.x / 2; i > 32; i >>= 1)
    {
        if (tid < i)
        {
            shared_mem[tid] += shared_mem[tid + i];
            __syncthreads();
        }
    }

    if (tid < 32)
        warpReduce(shared_mem, tid);
    if (tid == 0)
        d_output[blockIdx.x] = shared_mem[tid];
}

__global__ void reduce6(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = 2 * blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid] + d_input[global_tid + blockDim.x];
    __syncthreads();

    if(THREAD_PER_BLOCK >= 512){
        if(tid < 256) shared_mem[tid] += shared_mem[tid + 256];
        __syncthreads();
    }

    if(THREAD_PER_BLOCK >= 256){
        if(tid < 128) shared_mem[tid] += shared_mem[tid + 128];
        __syncthreads();
    }

    if(THREAD_PER_BLOCK >= 128){
        if(tid < 64) shared_mem[tid] += shared_mem[tid + 64];
        __syncthreads();
    }

    if (tid < 32) warpReduce(shared_mem, tid);

    if (tid == 0) d_output[blockIdx.x] = shared_mem[tid];
}

bool check(float *output, float *res, int n)
{
    for (int i = 0; i < n; ++i)
    {
        if (abs(output[i] - res[i]) > 0.0005)
            return false;
    }
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    // Use cudaMalloc to alloc memory on gpu. Using double ptr for DIRECTLY modify the original d_input(make this address to point to the alloc'd gpu mem) more or less like using inference.
    hipMalloc((void **)&d_input, N * sizeof(N));

    int block_num = N / THREAD_PER_BLOCK;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));

    float *res = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; ++i)
    {
        input[i] = drand48();
    }

    // Using cpu compute naive reduce result
    for (int i = 0; i < block_num; ++i)
    {
        for (int j = 0; j < THREAD_PER_BLOCK; ++j)
        {
            res[i] += input[i * THREAD_PER_BLOCK + j];
        }
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    reduce0<<<Grid, Block>>>(d_input, d_output);
    reduce1<<<Grid, Block>>>(d_input, d_output);
    reduce2<<<Grid, Block>>>(d_input, d_output);
    reduce3<<<Grid, Block>>>(d_input, d_output);
    reduce5<<<Grid, Block>>>(d_input, d_output);
    reduce6<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, res, block_num))
        printf("Correct!\n");
    else
    {
        printf("Incorrect!\n");
        for (int i = 0; i < block_num; ++i)
        {
            if (output[i] != res[i])
            {
                printf("Incorrect element: %lf (exp: %lf) in index %i\n", output[i], res[i], i);
            }
        }
    }

    hipFree(d_input);
    hipFree(d_output);
}