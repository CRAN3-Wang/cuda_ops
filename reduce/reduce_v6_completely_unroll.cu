#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__device__ void warpReduce(volatile float *cache, unsigned int tid)
{
    if (THREAD_PER_BLOCK > 64)
        cache[tid] += cache[tid + 32];
    if (THREAD_PER_BLOCK > 32)
        cache[tid] += cache[tid + 16];
    if (THREAD_PER_BLOCK > 16)
        cache[tid] += cache[tid + 8];
    if (THREAD_PER_BLOCK > 8)
        cache[tid] += cache[tid + 4];
    if (THREAD_PER_BLOCK > 4)
        cache[tid] += cache[tid + 2];
    if (THREAD_PER_BLOCK > 2)
        cache[tid] += cache[tid + 1];
}

__global__ void reduce6(float *d_input, float *d_output)
{
    __shared__ float shared_mem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int global_tid = 2 * blockIdx.x * blockDim.x + tid;

    shared_mem[tid] = d_input[global_tid] + d_input[global_tid + blockDim.x];
    __syncthreads();

    // Unroll all the iterations by #pragma unroll or manually unroll
    // for (unsigned int i = blockDim.x / 2; i > 32; i >>= 1)
    // {
    //     if (tid < i)
    //     {
    //         shared_mem[tid] += shared_mem[tid + i];
    //         __syncthreads();
    //     }
    // }

    if (THREAD_PER_BLOCK >= 512)
    {
        if (tid < 256)
            shared_mem[tid] += shared_mem[tid + 256];
        __syncthreads();
    }

    if (THREAD_PER_BLOCK >= 256)
    {
        if (tid < 128)
            shared_mem[tid] += shared_mem[tid + 128];
        __syncthreads();
    }

    if (THREAD_PER_BLOCK >= 128)
    {
        if (tid < 64)
            shared_mem[tid] += shared_mem[tid + 64];
        __syncthreads();
    }

    if (tid < 32)
        warpReduce(shared_mem, tid);

    if (tid == 0)
        d_output[blockIdx.x] = shared_mem[tid];
}

bool check(float *output, float *res, int n)
{
    for (int i = 0; i < n; ++i)
    {
        if (abs(output[i] - res[i]) > 0.0005)
            return false;
    }
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    // Use cudaMalloc to alloc memory on gpu. Using double ptr for DIRECTLY modify the original d_input(make this address to point to the alloc'd gpu mem) more or less like using inference.
    hipMalloc((void **)&d_input, N * sizeof(N));

    // We will add one epoch when loading, conseqeuntly the block num reduced by half.
    int block_num = N / THREAD_PER_BLOCK / 2;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));

    float *res = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; ++i)
    {
        input[i] = drand48();
    }

    // Using cpu compute naive reduce result
    for (int i = 0; i < block_num; ++i)
    {
        for (int j = 0; j < 2 * THREAD_PER_BLOCK; ++j)
        {
            res[i] += input[i * 2 * THREAD_PER_BLOCK + j];
        }
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    reduce6<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, res, block_num))
        printf("Correct!\n");
    else
    {
        printf("Incorrect!\n");
        for (int i = 0; i < block_num; ++i)
        {
            if (output[i] != res[i])
            {
                printf("Incorrect element: %lf (exp: %lf) in index %i\n", output[i], res[i], i);
            }
        }
    }

    hipFree(d_input);
    hipFree(d_output);
}