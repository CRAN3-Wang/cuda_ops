#include "sgemv_utils.hpp"
#include "sgemv_v0_32.cuh"
#include "sgemv_v1_float4.cuh"
#include "sgemv_v2_16.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main()
{
    int m = 256;
    int n = 16;

    size_t memsize_A = m * n * sizeof(float);
    size_t memsize_x = n * sizeof(float);
    size_t memsize_y = m * sizeof(float);

    float *h_A = (float *)malloc(memsize_A);
    float *h_x = (float *)malloc(memsize_x);
    float *h_mysgemv_y = (float *)malloc(memsize_y);
    float *h_cublas_y = (float *)malloc(memsize_y);

    randomMatrix(m, n, h_A);
    randomVector(n, h_x);

    float *d_A;
    float *d_x;
    float *d_mysgemv_y;
    float *d_cublas_y;

    hipMalloc((void **)&d_A, memsize_A);
    hipMalloc((void **)&d_x, memsize_x);
    hipMalloc((void **)&d_mysgemv_y, memsize_y);
    hipMalloc((void **)&d_cublas_y, memsize_y);

    hipMemcpy(d_A, h_A, memsize_A, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, memsize_x, hipMemcpyHostToDevice);

    // {
    //     dim3 Grid(m / 4);
    //     dim3 Block(32, 4);
    //     sgemv0<<<Grid, Block>>>(d_A, d_x, d_mysgemv_y, m, n);
    // }

    // {
    //     dim3 Grid(m / 4);
    //     dim3 Block(32, 4);
    //     sgemv1<<<Grid, Block>>>(d_A, d_x, d_mysgemv_y, m, n);
    // }

    {
        dim3 Grid(m / 8);
        dim3 Block(32, 4);
        sgemv2<2><<<Grid, Block>>>(d_A, d_x, d_mysgemv_y, m, n);
    }
    hipMemcpy(h_mysgemv_y, d_mysgemv_y, memsize_y, hipMemcpyDeviceToHost);

    cublas_sgemv(d_A, d_x, d_cublas_y, m, n);
    hipMemcpy(h_cublas_y, d_cublas_y, memsize_y, hipMemcpyDeviceToHost);

    check(h_mysgemv_y, h_cublas_y, m);

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_mysgemv_y);
    hipFree(d_cublas_y);
    free(h_A);
    free(h_x);
    free(h_mysgemv_y);
    free(h_cublas_y);

    return 0;
}